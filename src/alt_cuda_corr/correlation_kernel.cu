#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>


#define BLOCK_H 4
#define BLOCK_W 8
#define BLOCK_HW BLOCK_H * BLOCK_W
#define CHANNEL_STRIDE 32


__forceinline__ __device__
bool within_bounds(int h, int w, int H, int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}

template <typename scalar_t>
__global__ void corr_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap2,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> coords,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> corr,
    int r)
{
  const int b = blockIdx.x;
  const int h0 = blockIdx.y * blockDim.x;
  const int w0 = blockIdx.z * blockDim.y;
  const int tid = threadIdx.x * blockDim.y + threadIdx.y;

  const int H1 = fmap1.size(1);
  const int W1 = fmap1.size(2);
  const int H2 = fmap2.size(1);
  const int W2 = fmap2.size(2);
  const int N = coords.size(1);
  const int C = fmap1.size(3);

  __shared__ scalar_t f1[CHANNEL_STRIDE][BLOCK_HW+1];
  __shared__ scalar_t f2[CHANNEL_STRIDE][BLOCK_HW+1];
  __shared__ scalar_t x2s[BLOCK_HW];
  __shared__ scalar_t y2s[BLOCK_HW];

  for (int c=0; c<C; c+=CHANNEL_STRIDE) {
    for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
      int k1 = k + tid / CHANNEL_STRIDE;
      int h1 = h0 + k1 / BLOCK_W;
      int w1 = w0 + k1 % BLOCK_W;
      int c1 = tid % CHANNEL_STRIDE;

      auto fptr = fmap1[b][h1][w1];
      if (within_bounds(h1, w1, H1, W1))
        f1[c1][k1] = fptr[c+c1];
      else
        f1[c1][k1] = 0.0;
    }

    __syncthreads();

    for (int n=0; n<N; n++) {
      int h1 = h0 + threadIdx.x;
      int w1 = w0 + threadIdx.y;
      if (within_bounds(h1, w1, H1, W1)) {
        x2s[tid] = coords[b][n][h1][w1][0];
        y2s[tid] = coords[b][n][h1][w1][1];
      }

      scalar_t dx = x2s[tid] - floor(x2s[tid]);
      scalar_t dy = y2s[tid] - floor(y2s[tid]);

      int rd = 2*r + 1;
      for (int iy=0; iy<rd+1; iy++) {
        for (int ix=0; ix<rd+1; ix++) {
          for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
            int k1 = k + tid / CHANNEL_STRIDE;
            int h2 = static_cast<int>(floor(y2s[k1]))-r+iy;
            int w2 = static_cast<int>(floor(x2s[k1]))-r+ix;
            int c2 = tid % CHANNEL_STRIDE;

            auto fptr = fmap2[b][h2][w2];
            if (within_bounds(h2, w2, H2, W2))
              f2[c2][k1] = fptr[c+c2];
            else
              f2[c2][k1] = 0.0;
          }

          __syncthreads();
      
          scalar_t s = 0.0;
          for (int k=0; k<CHANNEL_STRIDE; k++)
            s += f1[k][tid] * f2[k][tid];

          int ix_nw = H1*W1*((iy-1) + rd*(ix-1));
          int ix_ne = H1*W1*((iy-1) + rd*ix);
          int ix_sw = H1*W1*(iy + rd*(ix-1));
          int ix_se = H1*W1*(iy + rd*ix);

          scalar_t nw = s * (dy) * (dx);
          scalar_t ne = s * (dy) * (1-dx);
          scalar_t sw = s * (1-dy) * (dx);
          scalar_t se = s * (1-dy) * (1-dx);

          scalar_t* corr_ptr = &corr[b][n][0][h1][w1];

          if (iy > 0 && ix > 0 && within_bounds(h1, w1, H1, W1))
            *(corr_ptr + ix_nw) += nw;

          if (iy > 0 && ix < rd && within_bounds(h1, w1, H1, W1))
            *(corr_ptr + ix_ne) += ne;

          if (iy < rd && ix > 0 && within_bounds(h1, w1, H1, W1))
            *(corr_ptr + ix_sw) += sw;

          if (iy < rd && ix < rd && within_bounds(h1, w1, H1, W1))
            *(corr_ptr + ix_se) += se;
        }
      } 
    }
  }
}


template <typename scalar_t>
__global__ void corr_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap1,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap2,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> coords,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> corr_grad,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap1_grad,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> fmap2_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> coords_grad,
    int r)
{

  const int b = blockIdx.x;
  const int h0 = blockIdx.y * blockDim.x;
  const int w0 = blockIdx.z * blockDim.y;
  const int tid = threadIdx.x * blockDim.y + threadIdx.y;

  const int H1 = fmap1.size(1);
  const int W1 = fmap1.size(2);
  const int H2 = fmap2.size(1);
  const int W2 = fmap2.size(2);
  const int N = coords.size(1);
  const int C = fmap1.size(3);

  __shared__ scalar_t f1[CHANNEL_STRIDE][BLOCK_HW+1];
  __shared__ scalar_t f2[CHANNEL_STRIDE][BLOCK_HW+1];

  __shared__ scalar_t f1_grad[CHANNEL_STRIDE][BLOCK_HW+1];
  __shared__ scalar_t f2_grad[CHANNEL_STRIDE][BLOCK_HW+1];

  __shared__ scalar_t x2s[BLOCK_HW];
  __shared__ scalar_t y2s[BLOCK_HW];

  for (int c=0; c<C; c+=CHANNEL_STRIDE) {

    for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
      int k1 = k + tid / CHANNEL_STRIDE;
      int h1 = h0 + k1 / BLOCK_W;
      int w1 = w0 + k1 % BLOCK_W;
      int c1 = tid % CHANNEL_STRIDE;

      auto fptr = fmap1[b][h1][w1];
      if (within_bounds(h1, w1, H1, W1))
        f1[c1][k1] = fptr[c+c1];
      else
        f1[c1][k1] = 0.0;

      f1_grad[c1][k1] = 0.0;
    }

    __syncthreads();

    int h1 = h0 + threadIdx.x;
    int w1 = w0 + threadIdx.y;

    for (int n=0; n<N; n++) {  
      x2s[tid] = coords[b][n][h1][w1][0];
      y2s[tid] = coords[b][n][h1][w1][1];

      scalar_t dx = x2s[tid] - floor(x2s[tid]);
      scalar_t dy = y2s[tid] - floor(y2s[tid]);

      int rd = 2*r + 1;
      for (int iy=0; iy<rd+1; iy++) {
        for (int ix=0; ix<rd+1; ix++) {
          for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
            int k1 = k + tid / CHANNEL_STRIDE;
            int h2 = static_cast<int>(floor(y2s[k1]))-r+iy;
            int w2 = static_cast<int>(floor(x2s[k1]))-r+ix;
            int c2 = tid % CHANNEL_STRIDE;

            auto fptr = fmap2[b][h2][w2];
            if (within_bounds(h2, w2, H2, W2))
              f2[c2][k1] = fptr[c+c2];
            else
              f2[c2][k1] = 0.0;

            f2_grad[c2][k1] = 0.0;
          }

          __syncthreads();
      
          const scalar_t* grad_ptr = &corr_grad[b][n][0][h1][w1];
          scalar_t g = 0.0;

          int ix_nw = H1*W1*((iy-1) + rd*(ix-1));
          int ix_ne = H1*W1*((iy-1) + rd*ix);
          int ix_sw = H1*W1*(iy + rd*(ix-1));
          int ix_se = H1*W1*(iy + rd*ix);

          if (iy > 0 && ix > 0 && within_bounds(h1, w1, H1, W1))
            g +=  *(grad_ptr + ix_nw) * dy * dx;

          if (iy > 0 && ix < rd && within_bounds(h1, w1, H1, W1))
            g += *(grad_ptr + ix_ne) * dy * (1-dx);

          if (iy < rd && ix > 0 && within_bounds(h1, w1, H1, W1))
            g += *(grad_ptr + ix_sw) * (1-dy) * dx;

          if (iy < rd && ix < rd && within_bounds(h1, w1, H1, W1))
            g += *(grad_ptr + ix_se) * (1-dy) * (1-dx);
            
          for (int k=0; k<CHANNEL_STRIDE; k++) {
            f1_grad[k][tid] += g * f2[k][tid];
            f2_grad[k][tid] += g * f1[k][tid];
          }

          for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
            int k1 = k + tid / CHANNEL_STRIDE;
            int h2 = static_cast<int>(floor(y2s[k1]))-r+iy;
            int w2 = static_cast<int>(floor(x2s[k1]))-r+ix;
            int c2 = tid % CHANNEL_STRIDE;

            scalar_t* fptr = &fmap2_grad[b][h2][w2][0];
            if (within_bounds(h2, w2, H2, W2))
              atomicAdd(fptr+c+c2, f2_grad[c2][k1]);
          }
        }
      } 
    }
    __syncthreads();


    for (int k=0; k<BLOCK_HW; k+=BLOCK_HW/CHANNEL_STRIDE) {
      int k1 = k + tid / CHANNEL_STRIDE;
      int h1 = h0 + k1 / BLOCK_W;
      int w1 = w0 + k1 % BLOCK_W;
      int c1 = tid % CHANNEL_STRIDE;

      scalar_t* fptr = &fmap1_grad[b][h1][w1][0];
      if (within_bounds(h1, w1, H1, W1))
        fptr[c+c1] += f1_grad[c1][k1];
    }
  }
}



std::vector<torch::Tensor> corr_cuda_forward(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  int radius)
{
  const auto B = coords.size(0);
  const auto N = coords.size(1);
  const auto H = coords.size(2);
  const auto W = coords.size(3);

  const auto rd = 2 * radius + 1;
  auto opts = fmap1.options();
  auto corr = torch::zeros({B, N, rd*rd, H, W}, opts);
  
  const dim3 blocks(B, (H+BLOCK_H-1)/BLOCK_H, (W+BLOCK_W-1)/BLOCK_W);
  const dim3 threads(BLOCK_H, BLOCK_W);

  corr_forward_kernel<float><<<blocks, threads>>>(
    fmap1.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    fmap2.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    corr.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    radius);

  return {corr};
}

std::vector<torch::Tensor> corr_cuda_backward(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  torch::Tensor corr_grad,
  int radius)
{
  const auto B = coords.size(0);
  const auto N = coords.size(1);

  const auto H1 = fmap1.size(1);
  const auto W1 = fmap1.size(2);
  const auto H2 = fmap2.size(1);
  const auto W2 = fmap2.size(2);
  const auto C = fmap1.size(3);

  auto opts = fmap1.options();
  auto fmap1_grad = torch::zeros({B, H1, W1, C}, opts);
  auto fmap2_grad = torch::zeros({B, H2, W2, C}, opts);
  auto coords_grad = torch::zeros({B, N, H1, W1, 2}, opts);
    
  const dim3 blocks(B, (H1+BLOCK_H-1)/BLOCK_H, (W1+BLOCK_W-1)/BLOCK_W);
  const dim3 threads(BLOCK_H, BLOCK_W);


  corr_backward_kernel<float><<<blocks, threads>>>(
    fmap1.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    fmap2.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    corr_grad.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    fmap1_grad.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    fmap2_grad.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    coords_grad.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
    radius);

  return {fmap1_grad, fmap2_grad, coords_grad};
}